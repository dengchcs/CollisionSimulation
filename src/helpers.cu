﻿#include <iostream>

#include "hip/hip_runtime_api.h"
#include "helpers.cuh"

void check(hipError_t error, const char *name) {
    if (error != hipSuccess) {
        std::cerr << name << " " << hipGetErrorString(error) << '\n';
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}
